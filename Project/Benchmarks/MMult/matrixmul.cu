#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
//#include "kernel.h"
//#include "kernel.cu"
#include "dev_array.h"
#include <math.h>
using namespace std;
struct timeval startS, endS, startP, endP;
float runtimeS, runtimeP;
void matrixMultiplication(float *A, float *B, float *C, int N);

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {

	int ROW = blockIdx.y*blockDim.y + threadIdx.y;
	int COL = blockIdx.x*blockDim.x + threadIdx.x;

	float tmpSum = 0;

	if (ROW < N && COL < N) {
		// each thread computes one element of the block sub-matrix
		for (int i = 0; i < N; i++) {
			tmpSum += A[ROW * N + i] * B[i * N + COL];
		}
	}
	C[ROW * N + COL] = tmpSum;
}


void matrixMultiplication(float *A, float *B, float *C, int N) {

	// declare the number of blocks per grid and the number of threads per block
	// use 1 to 512 threads per block
	dim3 threadsPerBlock(N, N);
	dim3 blocksPerGrid(1, 1);
	if (N*N > 512) {
		threadsPerBlock.x = 512;
		threadsPerBlock.y = 512;
		blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));
	}

	matrixMultiplicationKernel << <blocksPerGrid, threadsPerBlock >> > (A, B, C, N);
}






int main(int argc, char* argv[])
{
	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices

	int N = atoi(argv[1]);
	//int N = 16;
	int SIZE = N * N;
	gettimeofday(&startP, NULL);

	// Allocate memory on the host
	vector<float> h_A(SIZE);
	vector<float> h_B(SIZE);
	vector<float> h_C(SIZE);

	// Initialize matrices on the host
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			h_A[i*N + j] = sin(i);
			h_B[i*N + j] = cos(j);
		}
	}

	// Allocate memory on the device
	dev_array<float> d_A(SIZE);
	dev_array<float> d_B(SIZE);
	dev_array<float> d_C(SIZE);

	d_A.set(&h_A[0], SIZE);
	d_B.set(&h_B[0], SIZE);

	matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
	hipDeviceSynchronize();

	d_C.get(&h_C[0], SIZE);
	hipDeviceSynchronize();

	float *cpu_C;
	cpu_C = new float[SIZE];
	gettimeofday(&endP, NULL);

	//
	// Calculate the runtime.
	//
	runtimeP = ((endP.tv_sec - startP.tv_sec) * 1000.0) + ((endP.tv_usec - startP.tv_usec) / 1000.0);
	printf("Processing Time Parallel: %4.4f milliseconds\n", runtimeP);

	gettimeofday(&startS, NULL);

	// Now do the matrix multiplication on the CPU
	float sum;
	for (int row = 0; row < N; row++) {
		for (int col = 0; col < N; col++) {
			sum = 0.f;
			for (int n = 0; n < N; n++) {
				sum += h_A[row*N + n] * h_B[n*N + col];
			}
			cpu_C[row*N + col] = sum;
		}
	}
	gettimeofday(&endS, NULL);
	double err = 0;
	// Check the result and make sure it is correct
	for (int ROW = 0; ROW < N; ROW++) {
		for (int COL = 0; COL < N; COL++) {
			err += cpu_C[ROW * N + COL] - h_C[ROW * N + COL];
		}
	}


	//
	// Calculate the runtime.
	//
	runtimeS = ((endS.tv_sec - startS.tv_sec) * 1000.0) + ((endS.tv_usec - startS.tv_usec) / 1000.0);
	printf("Processing Time Serial: %4.4f milliseconds\n", runtimeS);

	cout << "Error: " << err << endl;

	return 0;
}


